#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <numeric>
#include <iostream>

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

double random_double(void)
{

  return static_cast<double>(rand()) / RAND_MAX;
}


// Part 1 of 6: implement the kernel
__global__ void block_sum(const double *input,
                          double *per_block_results,
                          const size_t n)
{
  //fill me
  __shared__ double sdata[];



}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(void)
{
  // create array of 256ki elements
  const int num_elements = 1<<18;
  srand(time(NULL));
  // generate random input on the host
  std::vector<double> h_input(num_elements);
  for(int i = 0; i < h_input.size(); ++i)
  {
    h_input[i] = random_double();
  }

  const double host_result = std::accumulate(h_input.begin(), h_input.end(), 0.0f);
  std::cerr << "Host sum: " << host_result << std::endl;

  //Part 1 of 6: move input to device memory
  double *d_input = 0;

  // Part 1 of 6: allocate the partial sums: How much space does it need?
  double *d_partial_sums_and_total = 0;

  // Part 1 of 6: launch one kernel to compute, per-block, a partial sum. How much shared memory does it need?
  block_sum<<<num_blocks,block_size>>>(d_input, d_partial_sums_and_total, num_elements);

  // Part 1 of 6: compute the sum of the partial sums
  block_sum<<<>>>();

  // Part 1 of 6: copy the result back to the host
  double device_result = 0;

  std::cout << "Device sum: " << device_result << std::endl;

  // Part 1 of 6: deallocate device memory


  return 0;
}
